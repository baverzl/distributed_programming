#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define BLK_SIZE 100
#define MAX_NUM_THREADS_PER_BLK 1024

const int N = 1e2;

__global__ void reduce0(int *g_idata, int *g_odata) {
	__shared__ int sdata[MAX_NUM_THREADS_PER_BLK];

	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + tid;

//	printf("%d %d %d\n", blockIdx.x, blockDim.x, tid);

	sdata[tid] = g_idata[i];
	__syncthreads();
	
	// do reduction in shared mem
	// sequential accessing
	for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if(tid < s) {
			sdata[tid] = (sdata[tid] < sdata[tid + s])? sdata[tid + s] : sdata[tid];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if(tid == 0) g_odata[blockIdx.x] = sdata[0];
}

int main(void)
{
	int host_idata[N];
	int host_odata[MAX_NUM_THREADS_PER_BLK];
	int *dev_idata, *dev_odata;

	hipMalloc((void **) &dev_idata, sizeof(int) * N);
	hipMalloc((void **) &dev_odata, sizeof(int) * MAX_NUM_THREADS_PER_BLK);

	srand(time(NULL));	

	for(int i = 0; i < N; i++) {
		host_idata[i] = rand() % 128;		
	}

	hipMemcpy(dev_idata, host_idata, N * sizeof(int), hipMemcpyHostToDevice);
	
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	reduce0<<<BLK_SIZE, MAX_NUM_THREADS_PER_BLK>>> (dev_idata, dev_odata);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&time, start, stop);

	printf("Time for the kernel: %f\n", time);

	hipDeviceSynchronize();

	hipMemcpy(host_odata, dev_odata, BLK_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	int max = 0;
	for(int i = 0; i < BLK_SIZE; i++)
		if(max < host_odata[i])
			max = host_odata[i];

	printf("reduction max : %d\n", max);

	return 0;

}

